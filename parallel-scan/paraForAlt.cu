#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <cmath>

const int N = 1256;
const int BLOCK_SIZE = 256;

__global__ void pairs_trading_kernel(const double* stock1_prices, const double* stock2_prices, int* check, int size) {
    __shared__ double spread[N];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < N; i += stride) {
        spread[i] = stock1_prices[i] - stock2_prices[i];
    }

    __syncthreads();

    for (int i = idx + N; i < size; i += stride) {
        double sum = 0.0;
        double sq_sum = 0.0;

#pragma unroll
        for (int j = 0; j < 8; ++j) {
            double val = spread[i - N + j];
            sum += val;
            sq_sum += val * val;
        }

        double mean = sum / N;
        double stddev = sqrt(sq_sum / N - mean * mean);
        double current_spread = stock1_prices[i] - stock2_prices[i];
        double z_score = (current_spread - mean) / stddev;

        if (z_score > 1.0) {
            atomicAdd(&check[0], 1);  // Long and Short
        } else if (z_score < -1.0) {
            atomicAdd(&check[1], 1);  // Short and Long
        } else if (fabs(z_score) < 0.8) {
            atomicAdd(&check[2], 1);  // Close positions
        } else {
            atomicAdd(&check[3], 1);  // No signal
        }
    }
}

void pairs_trading_strategy_cuda(const std::vector<double>& stock1_prices, const std::vector<double>& stock2_prices) {
    int size = stock1_prices.size();

    thrust::device_vector<double> d_stock1_prices = stock1_prices;
    thrust::device_vector<double> d_stock2_prices = stock2_prices;
    thrust::device_vector<int> d_check(4, 0);

    int grid_size = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    pairs_trading_kernel<<<grid_size, BLOCK_SIZE>>>(
            thrust::raw_pointer_cast(d_stock1_prices.data()),
            thrust::raw_pointer_cast(d_stock2_prices.data()),
            thrust::raw_pointer_cast(d_check.data()),
            size
    );

    hipDeviceSynchronize();

    std::vector<int> check(4);
    thrust::copy(d_check.begin(), d_check.end(), check.begin());

    std::cout << check[0] << ":" << check[1] << ":" << check[2] << ":" << check[3] << std::endl;
}