#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <vector>
#include <stdlib.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

#include "kernels.cuh"
#include "utils.h"
#include "scan.cuh"

#define checkCudaError(o, l) _checkCudaError(o, l, __func__)

int THREADS_PER_BLOCK = 512;
int ELEMENTS_PER_BLOCK = THREADS_PER_BLOCK * 2;

using namespace std;

long sequential_scan(double* output, double* input, int length) {
	long start_time = get_nanos();

	output[0] = 0; // since this is a prescan, not a scan
	for (int j = 1; j < length; ++j)
	{
		output[j] = input[j - 1] + output[j - 1];
	}

	long end_time = get_nanos();
	return end_time - start_time;
}

/*float blockscan(int *output, int *input, int length, bool bcao){
	int *d_out, *d_in;
	const int arraySize = length * sizeof(int);

	hipMalloc((void **)&d_out, arraySize);
	hipMalloc((void **)&d_in, arraySize);
	hipMemcpy(d_out, output, arraySize, hipMemcpyHostToDevice);
	hipMemcpy(d_in, input, arraySize, hipMemcpyHostToDevice);

	// start timer
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	int powerOfTwo = nextPowerOfTwo(length);
	if (bcao) {
		prescan_arbitrary<<<1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int)>>>(d_out, d_in, length, powerOfTwo);
	}
	else {
		prescan_arbitrary_unoptimized<<<1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int)>>>(d_out, d_in, length, powerOfTwo);
	}
	
	// end timer
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipMemcpy(output, d_out, arraySize, hipMemcpyDeviceToHost);

	hipFree(d_out);
	hipFree(d_in);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return elapsedTime;
}*/

float scan(double *output, double *input, int length, bool bcao) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
	double *d_out, *d_in;
	const int arraySize = length * sizeof(double);

	hipMalloc((void **)&d_out, arraySize);
	hipMalloc((void **)&d_in, arraySize);
	hipMemcpy(d_out, output, arraySize, hipMemcpyHostToDevice);
	hipMemcpy(d_in, input, arraySize, hipMemcpyHostToDevice);

	/*// start timer
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);*/

	if (length > ELEMENTS_PER_BLOCK) {
		scanLargeDeviceArray(d_out, d_in, length, bcao);
	}
	else {
		scanSmallDeviceArray(d_out, d_in, length, bcao);
	}

	// end timer
	/*hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, start, stop);*/

	hipMemcpy(input, d_out, arraySize, hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsedTime = 0;
    hipEventElapsedTime(&elapsedTime, start, stop);

	hipFree(d_out);
	hipFree(d_in);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return elapsedTime;
}


void scanLargeDeviceArray(double *d_out, double *d_in, int length, bool bcao) {
	int remainder = length % (ELEMENTS_PER_BLOCK);
	if (remainder == 0) {
		scanLargeEvenDeviceArray(d_out, d_in, length, bcao);
	}
	else {
		// perform a large scan on a compatible multiple of elements
		int lengthMultiple = length - remainder;
		scanLargeEvenDeviceArray(d_out, d_in, lengthMultiple, bcao);

		// scan the remaining elements and add the (inclusive) last element of the large scan to this
		double *startOfOutputArray = &(d_out[lengthMultiple]);
		scanSmallDeviceArray(startOfOutputArray, &(d_in[lengthMultiple]), remainder, bcao);

		add<<<1, remainder>>>(startOfOutputArray, remainder, &(d_in[lengthMultiple - 1]), &(d_out[lengthMultiple - 1]));
	}
}

void scanSmallDeviceArray(double *d_out, double *d_in, int length, bool bcao) {
	int powerOfTwo = nextPowerOfTwo(length);

	if (bcao) {
		prescan_arbitrary << <1, (length + 1) / 2, 2 * powerOfTwo * sizeof(double) >> >(d_out, d_in, length, powerOfTwo);
	}
	else {
		prescan_arbitrary_unoptimized<< <1, (length + 1) / 2, 2 * powerOfTwo * sizeof(double) >> >(d_out, d_in, length, powerOfTwo);
	}
}

void scanLargeEvenDeviceArray(double *d_out, double *d_in, int length, bool bcao) {
	const int blocks = length / ELEMENTS_PER_BLOCK;
	const double sharedMemArraySize = ELEMENTS_PER_BLOCK * sizeof(double);

	double *d_sums, *d_incr;
	hipMalloc((void **)&d_sums, blocks * sizeof(double));
	hipMalloc((void **)&d_incr, blocks * sizeof(double));

	if (bcao) {
		prescan_large<<<blocks, THREADS_PER_BLOCK, 2 * sharedMemArraySize>>>(d_out, d_in, ELEMENTS_PER_BLOCK, d_sums);
	}
	else {
		prescan_large_unoptimized<<<blocks, THREADS_PER_BLOCK, 2 * sharedMemArraySize>>>(d_out, d_in, ELEMENTS_PER_BLOCK, d_sums);
	}

	const int sumsArrThreadsNeeded = (blocks + 1) / 2;
	if (sumsArrThreadsNeeded > THREADS_PER_BLOCK) {
		// perform a large scan on the sums arr
		scanLargeDeviceArray(d_incr, d_sums, blocks, bcao);
	}
	else {
		// only need one block to scan sums arr so can use small scan
		scanSmallDeviceArray(d_incr, d_sums, blocks, bcao);
	}

	add<<<blocks, ELEMENTS_PER_BLOCK>>>(d_out, ELEMENTS_PER_BLOCK, d_incr);

	hipFree(d_sums);
	hipFree(d_incr);
}

__global__ void parallelized_zscore_calculation(
        const double *stock1_prices,
        const double *stock2_prices,
        const double *spread_sum,
        const double *spread_sq_sum,
        int *check,
        int N,
        size_t size) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("idx:%d\n", idx);
    //if (idx >= size) return;
    //if(idx >= 1247)printf("idx:%d\n", idx);
    if (idx >= size - N) return;
    //if(idx >= 1247)printf("idx:%d\n", idx);

    int i = N + idx;
    //printf("i:%d\n", i);
    //if(i >= size)return;
    //printf("i:%d\n", i);
    const double mean = (spread_sum[i] - spread_sum[i-N])/ N;
    const double stddev = std::sqrt((spread_sq_sum[i] - spread_sq_sum[i-N])/ N - mean * mean);
    const double current_spread = stock1_prices[i] - stock2_prices[i];
    const double z_score = (current_spread - mean) / stddev;

    if (z_score > 1.0) {
        atomicAdd(&check[0], 1); // Long and Short
    } else if (z_score < -1.0) {
        atomicAdd(&check[1], 1); // Short and Long
    } else if (std::abs(z_score) < 0.8) {
        atomicAdd(&check[2], 1);  // Close positions
    } else {
        atomicAdd(&check[3], 1);  // No signal
    }
}

__global__ void parallelized_zscore_calculation1(
        const double *stock1_prices,
        const double *stock2_prices,
        const double *spread_sum,
        const double *spread_sq_sum,
        int *check,
        int N,
        size_t size) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("idx:%d\n", idx);
    //if (idx >= size) return;
    if (idx >= size - N - 1) return;


    int i = N + 1 + idx;
    //printf("i:%d\n", i);
    printf("stock1price:%d\n", stock1_prices[i]);
    const double mean = (spread_sum[i-1] - spread_sum[i-N-1])/ N;
    const double stddev = std::sqrt((spread_sq_sum[i-1] - spread_sq_sum[i-N-1])/ N - mean * mean);
    const double current_spread = stock1_prices[i] - stock2_prices[i];
    const double z_score = (current_spread - mean) / stddev;

    if (z_score > 1.0) {
        //atomicAdd(&check[0], 1); // Long and Short
    } else if (z_score < -1.0) {
        //atomicAdd(&check[1], 1); // Short and Long
    } else if (std::abs(z_score) < 0.8) {
        //atomicAdd(&check[2], 1);  // Close positions
    } else {
        //atomicAdd(&check[3], 1);  // No signal
    }
}

void calc_z(const std::vector<double>& stock1_prices, const std::vector<double>& stock2_prices,
            const std::vector<double>& spread_sum, const std::vector<double>& spread_sq_sum,
            std::vector<int>& check) {
    const int N = 8;
    double *d_stock1_prices, *d_stock2_prices, *d_spread_sum, *d_spread_sq_sum;
    int *d_check;

    hipMalloc((void**)&d_stock1_prices, stock1_prices.size() * sizeof(double));
    hipMalloc((void**)&d_stock2_prices, stock2_prices.size() * sizeof(double));
    hipMalloc((void**)&d_spread_sum, spread_sum.size() * sizeof(double));
    hipMalloc((void**)&d_spread_sq_sum, spread_sq_sum.size() * sizeof(double));
    hipMalloc((void**)&d_check, check.size() * sizeof(int)); // Assuming 'check' has size 4

// Data Transfer to the GPU
    hipMemcpy(d_stock1_prices, stock1_prices.data(), stock1_prices.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_stock2_prices, stock2_prices.data(), stock2_prices.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_spread_sum, spread_sum.data(), spread_sum.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_spread_sq_sum, spread_sq_sum.data(), spread_sq_sum.size() * sizeof(double), hipMemcpyHostToDevice);

    int threadsPerBlock = 512;
    //int numBlocks = (stock1_prices.size() + threadsPerBlock - 1) / threadsPerBlock;
    int numBlocks = (stock1_prices.size() - N - 1 + threadsPerBlock - 1) / threadsPerBlock;

    //printf("%d\n", numBlocks);

    parallelized_zscore_calculation1<<<numBlocks, threadsPerBlock >>>(d_stock1_prices, d_stock2_prices, d_spread_sum, d_spread_sq_sum, d_check, N, stock1_prices.size());

// Copy results back
    hipMemcpy(check.data(), d_check, check.size() * sizeof(int), hipMemcpyDeviceToHost);

// Print results
    //std::cout<<check[0]<<":"<<check[1]<<":"<<check[2]<<":"<<check[3]<<std::endl;
    printf("d_check[0]:%d || d_check[1]:%d || d_check[2]:%d || d_check[3]:%d \n", check[0], check[1], check[2], check[3]);
    hipFree(d_stock1_prices);
    hipFree(d_stock2_prices);
    hipFree(d_spread_sum);
    hipFree(d_spread_sq_sum);
    hipFree(d_check);
}

void calc_zz(const std::vector<double>& stock1_prices, const std::vector<double>& stock2_prices,
            double spread_sum[], double spread_sq_sum[],
            std::vector<int>& check, size_t spread_size) {
    const int N = 8;
    double *d_stock1_prices, *d_stock2_prices, *d_spread_sum, *d_spread_sq_sum;
    int *d_check;

    hipMalloc((void**)&d_stock1_prices, stock1_prices.size() * sizeof(double));
    hipMalloc((void**)&d_stock2_prices, stock2_prices.size() * sizeof(double));
    hipMalloc((void**)&d_spread_sum, spread_size * sizeof(double));
    hipMalloc((void**)&d_spread_sq_sum, spread_size * sizeof(double));
    hipMalloc((void**)&d_check, check.size() * sizeof(int)); // Assuming 'check' has size 4

// Data Transfer to the GPU
    hipMemcpy(d_stock1_prices, stock1_prices.data(), stock1_prices.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_stock2_prices, stock2_prices.data(), stock2_prices.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_spread_sum, spread_sum, spread_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_spread_sq_sum, spread_sq_sum, spread_size * sizeof(double), hipMemcpyHostToDevice);

    int threadsPerBlock = 512;

    int numBlocks = (stock1_prices.size() - N - 1 + threadsPerBlock - 1) / threadsPerBlock;
    //printf("numBlocks:%d\n", numBlocks);

    parallelized_zscore_calculation<<<numBlocks, threadsPerBlock >>>(d_stock1_prices, d_stock2_prices, d_spread_sum, d_spread_sq_sum, d_check, N, stock1_prices.size());

// Copy results back
    hipMemcpy(check.data(), d_check, check.size() * sizeof(int), hipMemcpyDeviceToHost);

// Print results
    //std::cout<<check[0]<<":"<<check[1]<<":"<<check[2]<<":"<<check[3]<<std::endl;
    printf("d_check[0]:%d || d_check[1]:%d || d_check[2]:%d || d_check[3]:%d \n", check[0], check[1], check[2], check[3]);
    hipFree(d_stock1_prices);
    hipFree(d_stock2_prices);
    hipFree(d_spread_sum);
    hipFree(d_spread_sq_sum);
    hipFree(d_check);
}

__global__ void para_fill(const double *stock1_prices,
                          const double *stock2_prices,
                          double *spread_sum,
                          double *spread_sq_sum,
                          size_t size){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= size)return;
    const double current_spread = stock1_prices[idx] - stock2_prices[idx];
    spread_sum[idx] = current_spread;
    spread_sq_sum[idx] = current_spread * current_spread;
}


void fillArrays(const std::vector<double>& stock1_prices, const std::vector<double>& stock2_prices,
                             double spread_sum[], double spread_sq_sum[], size_t spread_size){
    double *d_stock1_prices, *d_stock2_prices, *d_spread_sum, *d_spread_sq_sum;

    hipMalloc((void**)&d_stock1_prices, stock1_prices.size() * sizeof(double));
    hipMalloc((void**)&d_stock2_prices, stock2_prices.size() * sizeof(double));
    hipMalloc((void**)&d_spread_sum, spread_size * sizeof(double));
    hipMalloc((void**)&d_spread_sq_sum, spread_size * sizeof(double));

    hipMemcpy(d_stock1_prices, stock1_prices.data(), stock1_prices.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_stock2_prices, stock2_prices.data(), stock2_prices.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_spread_sum, spread_sum, spread_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_spread_sq_sum, spread_sq_sum, spread_size * sizeof(double), hipMemcpyHostToDevice);

    int threadsPerBlock = 512;

    int numBlocks = (stock1_prices.size() + threadsPerBlock - 1) / threadsPerBlock;

    para_fill<<<numBlocks, threadsPerBlock >>>(d_stock1_prices, d_stock2_prices, d_spread_sum, d_spread_sq_sum, spread_size);

    hipMemcpy(spread_sum, d_spread_sum, spread_size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(spread_sq_sum, d_spread_sq_sum, spread_size * sizeof(double), hipMemcpyDeviceToHost);


    hipFree(d_stock1_prices);
    hipFree(d_stock2_prices);
    hipFree(d_spread_sum);
    hipFree(d_spread_sq_sum);
}
